#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-matmul.cu - Dense matrix-matrix multiplication with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Dense matrix-matrix multiplication kernel with CUDA. Two versions
 * of the kernel are provided: one that does not use shared memory,
 * and one that does.
 *
 * Compile with:
 * nvcc cuda-matmul.cu -o cuda-matmul -lm
 *
 * Run with:
 * ./cuda-matmul [N]
 *
 ****************************************************************************/

#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>     /* for malloc() */
#include <math.h>       /* for fabsf()  */
#include <string.h>     /* for bzero()  */

#define BLKSIZE 16

/* Compute r = p * q, for square nxn matrices p, q, r; this version
   does not use shared memory. This kernel does not require that n is
   a multiple of BLKSIZE */
__global__ void matmul( const float *p, const float *q, float *r, int n )
{
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k;
    float val = 0.0;
    if ( i < n && j < n ) {
        for (k=0; k<n; k++) {
            val += p[i*n + k] * q[k*n + j];
        }
        r[i*n + j] = val;
    }
}

/* Compute r = p * q, for square nxn matrices p, q, r; this version
   uses shared memory. This kernel _requires_ that n is a multiple of
   BLKSIZE */
__global__ void matmulb( const float *p, const float *q, float *r, int n )
{
    __shared__ float local_p[BLKSIZE][BLKSIZE];
    __shared__ float local_q[BLKSIZE][BLKSIZE];
    const int bx = blockIdx.x;  const int by = blockIdx.y;
    const int tx = threadIdx.x; const int ty = threadIdx.y;
    const int i = by * BLKSIZE + ty;
    const int j = bx * BLKSIZE + tx;
    float v = 0.0; int m, k;
    for (m = 0; m < n; m += BLKSIZE) { /* loop over tiles */
        local_p[ty][tx] = p[i*n + (m + tx)];
        local_q[ty][tx] = q[(m + ty)*n + j];
        __syncthreads();
        for (k = 0; k < BLKSIZE; k++) { /* loop within tile */
            v += local_p[ty][k] * local_q[k][tx];
        }
        __syncthreads();
    }    
    r[i*n + j] = v; /* write back to global memory */
}


/* Initialize square matrix q */
void mat_init( float *q, int n )
{
    int i;
    for (i=0; i<n*n; i++) {
        q[i] = 1.0;
    }
}

int check_result( const float *r, int n ) 
{
    /* Check result */
    int i, j;
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            if (fabsf(r[i*n+j] - n) > 1e-5) {
                printf("Check failed: r[%d][%d] = %f, expected %f\n", i, j, r[i*n+j], (float)n);
                return 0;
            }
        }
    }
    printf("Check OK\n");
    return 1;
}

int main( int argc, char* argv[] ) 
{
    float *p, *q, *r;	          /* host copies of p, q, r */ 
    float *d_p, *d_q, *d_r;	  /* device copies of p, q, r */
    int N = 512;
    double tstart, tstop, tnoshared, tshared;

    if ( argc > 1 ) {
        N = atoi(argv[1]);
    }

    // FIXME: siamo sicuri?
    if ( N % BLKSIZE ) {
        printf("Error: the matrix size must be a multiple of %d\n", BLKSIZE);
        return -1;
    }

    dim3 block(BLKSIZE, BLKSIZE);
    dim3 grid((N+BLKSIZE-1)/BLKSIZE, (N+BLKSIZE-1)/BLKSIZE);
    const size_t size = N*N*sizeof(float);

    /* Allocate space for device copies of p, q, r */
    hipMalloc((void **)&d_p, size);
    hipMalloc((void **)&d_q, size);
    hipMalloc((void **)&d_r, size);

    /* Allocate space for host copies of p, q, r */
    p = (float*)malloc(size); mat_init(p, N);
    q = (float*)malloc(size); mat_init(q, N);
    r = (float*)malloc(size);

    /* Copy inputs to device */
    hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, size, hipMemcpyHostToDevice);

    printf("Matrix-Matrix multiplication (%dx%d)\n", N, N);

    /**
     ** Matrix-matrix multiply WITHOUT shared memory
     **/
    printf("No shared memory:\t");
    tstart = hpc_gettime();
    matmul<<<grid, block>>>(d_p, d_q, d_r, N);
    hipDeviceSynchronize();
    tstop = hpc_gettime();
    tnoshared = tstop - tstart;
    printf("%fs\n", tnoshared);
    /* Copy result back to host and check correctness */
    bzero(r, size); /* erase destination buffer, just in case... */
    hipMemcpy(r, d_r, size, hipMemcpyDeviceToHost);
    check_result(r, N);

    /* zero out r and d_r, to ensure that we don't read old results */
    hipMemset(d_r, 0, size);
    bzero(r, size);

    /**
     ** Matrix-matrix multiply WITH shared memory
     **/
    printf("Shared memory:\t\t");
    tstart = hpc_gettime();
    matmulb<<<grid, block>>>(d_p, d_q, d_r, N);
    hipDeviceSynchronize();
    tstop = hpc_gettime();
    tshared = tstop - tstart;
    printf("%fs (%.2fx speedup)\n", tshared, tnoshared / tshared);
    /* Copy result back to host and check correctness */
    hipMemcpy(r, d_r, size, hipMemcpyDeviceToHost);
    check_result(r, N);

    /* Cleanup */
    free(p); free(q); free(r);
    hipFree(d_p); hipFree(d_q); hipFree(d_r);
    return 0;
}
