/****************************************************************************
 *
 * cuda-vecadd1.cu - Sum two arrays with CUDA, using thread blocks
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Based on the example shown in the CUDA toolkit documentation
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/
 *
 * Compile with:
 * nvcc cuda-vecadd1.cu -o cuda-vecadd1
 *
 * Run with:
 * ./cuda-vecadd1
 *
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add( int *a, int *b, int *c )
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void vec_init( int *a, int n )
{
    int i;
    for (i=0; i<n; i++) {
        a[i] = i;
    }
}

#define N 512

int main( void ) 
{
    int *a, *b, *c;	          /* host copies of a, b, c */ 
    int *d_a, *d_b, *d_c;	  /* device copies of a, b, c */
    int i;
    const size_t size = N*sizeof(int);
    /* Allocate space for device copies of a, b, c */
    hipMalloc((void **)&d_a, size); 
    hipMalloc((void **)&d_b, size); 
    hipMalloc((void **)&d_c, size);
    /* Allocate space for host copies of a, b, c */
    a = (int*)malloc(size); vec_init(a, N);
    b = (int*)malloc(size); vec_init(b, N);
    c = (int*)malloc(size);
    /* Copy inputs to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    /* Launch add() kernel on GPU */
    printf("Adding %d elements\n", N);
    add<<<N,1>>>(d_a, d_b, d_c);
    /* Copy result back to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    /* Check results */
    for (i=0; i<N; i++) {
        if ( c[i] != a[i] + b[i] ) {
            printf("Error at index %d: a[%d]=%d, b[%d]=%d, c[%d]=%d\n",
                   i, i, a[i], i, b[i], i, c[i]);
            return -1;
        }
    }    
    printf("Test OK\n");
    /* Cleanup */
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
